#include "memtable.cuh"

void deleteMemtable(struct Memtable *mt) 
{
    hipFree(mt->d_sortedKeys); 
    hipFree(mt->d_sortedValuePointers); 
    hipFree(mt->d_sortedOperationIDs); 
}
